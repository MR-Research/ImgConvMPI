
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernelConv(double *a, double *b, double *c, int w, int h, int filtsize) {
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    int j = threadIdx.y+blockDim.y*blockIdx.y;
    int k,l, cont = 0, abspos;
    double sum = 0;    
    abspos = i+(j*w); 
    for (l = 0; l < filtsize; l++){              
        for (k = 0; k < filtsize; k++){                              
            sum += b[(abspos+l)+(k*w)] * a[cont];                      
            cont++;
        }
    }    
    c[i+(j*w)] = sum;      
}


float imgConvDevice(double *a, double *b, double *c, int filtsize, int w, int h) {
    double *aD, *bD, *cD; 
    hipEvent_t start, stop; 
    float t; 
    int wb = (int)floor((double)w/filtsize*1.0), hb = (int)floor((double)h/filtsize*1.0);
    dim3 bloques(wb,hb); 
    dim3 hilos(filtsize, filtsize); 
    printf("Grid (%d,%d)\n", wb-1, hb-1); 
    hipEventCreate(&start); hipEventCreate(&stop); 
    //cudaEventRecord(start, 0);
    hipMalloc(&aD, filtsize*filtsize*sizeof(double *)); 
    hipMalloc(&bD, w*h*sizeof(double *));
    hipMalloc(&cD, w*h*sizeof(double *));
    
    hipMemcpy(aD, a, filtsize*filtsize*sizeof(double *), hipMemcpyDefault); 
    hipMemcpy(bD, b, w*h*sizeof(double *), hipMemcpyDefault);

    hipEventRecord(start, 0);     
    kernelConv<<<bloques, hilos>>>(aD, bD, cD, w, h, filtsize);
    hipEventRecord(stop, 0); 

    hipMemcpy(c, cD, w*h*sizeof(double *), hipMemcpyDefault);


    hipFree(aD); hipFree(bD);
    //cudaEventRecord(stop, 0); 
    hipEventSynchronize(stop);


    hipEventElapsedTime(&t, start, stop); 
    hipEventDestroy(start); hipEventDestroy(stop); 
    return t; 
}


int main(int argc, char **argv) { // parameters: gpu deivce, image path, image width, image height, filter path, resulting image path, filter size
    
    if (argc < 6){
        printf("Error: number of agrguments incorrect.");
        return 0;
    }
    int d = atoi(argv[1]); 
    const char *imgpath = argv[2];
    const char *filterpath = argv[3];
    const char *imgrespath = argv[4];
    const char *imgw = argv[5];
    const char *imgh = argv[6];
    const char *filtsize = argv[7];      

    FILE *pf, *pimg, *pres;
    int i, j, fs, imw, imh, wc, hc;
    float t;
    double *a;
    double *b;
    double *c;
    double *tmp;  
    
    fs = atoi(filtsize);
    imw = atoi(imgw);
    imh = atoi(imgh);
    wc = imw - (fs-1);
    hc = imh - (fs-1);     
    hipSetDevice(d%3); 
    hipHostAlloc(&a, atoi(filtsize)*atoi(filtsize)*sizeof(double *), hipHostMallocDefault);      
    hipHostAlloc(&b, atoi(imgw)*atoi(imgh)*sizeof(double *), hipHostMallocDefault);  
    hipHostAlloc(&c, atoi(imgw)*atoi(imgh)*sizeof(double *), hipHostMallocDefault);  
    
    //Reading the filter from a file
    pf = fopen (filterpath, "r");
    if (pf == NULL){
        printf("Error loading filter\n");
        return 0;
    }   
    for(i = 0, tmp = a; i < atoi(filtsize)*atoi(filtsize); ++i){
            fscanf(pf, "%lf", tmp++);
    }
    fclose (pf); 
    //Image loading
    pimg = fopen (imgpath, "r");
    if (pimg == NULL){
        printf("Error loading image\n");
        return 0;
    }   
    for(i = 0, tmp = b; i < atoi(imgw)*atoi(imgh); ++i){
            fscanf(pimg, "%lf", tmp++);
    }
    fclose (pimg);    
    
    t = imgConvDevice(a, b, c, fs, imw, imh); 

    //Image writting
    pres = fopen (imgrespath, "w");
    if (pimg == NULL){
        printf("Error loading file to write\n");
        return 0;
    }   
    tmp = c;
    //for(i = 1, tmp = c; i < atoi(imgw)*atoi(imgh); ++i){            
    for (i = 0; i < hc; i++){    
        for (j = 0; j < wc; j++)        
            if (j == wc-1){
                fprintf(pres, "%lf", *tmp);
                fprintf(pres, "\n");     
                tmp = tmp + fs;   
            } else {
                fprintf(pres, "%lf,", *tmp);
                tmp++;
            }            
    }
    fclose (pres);     
    printf("Run time: %f s\n", t/1000);
    hipHostFree(a); 
    hipHostFree(b);
    hipHostFree(c);
}


